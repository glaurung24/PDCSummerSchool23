#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include "energy_storms_cuda.hpp"

namespace CUDA{

/* Function to get wall time */
double cp_Wtime(){
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}

void read_storm_files(int argc,
                    char* argv[], 
                    Storm* storms, 
                    const int& num_storms){
    for(int i=2; i<argc; i++ ) 
        storms[i-2] = read_storm_file( argv[i] );
}

struct above_threshold
{
    __host__ __device__
    bool operator()(float x)
    {
        return (x*((x>=0)*1-(x<0)) > THRESHOLD);
    }
};

void run_calculation(float* layer, const int& layer_size, Storm* storms, const int& num_storms,
                float* maximum,
                int* positions){
    //lookup table for prefactor 1/sqrt(distance)/layer_size
    //It needs to be 2*layer_size to fit 
    thrust::host_vector<float> look_up;
    look_up.reserve(2*layer_size);
    for(int i = 0; i<2*layer_size; i++){
        look_up.push_back(1.0f/sqrtf((float)abs(layer_size-i)+1)/(float)layer_size); //TODO this should be done on the gpu
    }
    thrust::device_vector<float> look_up_device = look_up;
    thrust::device_vector<float> layer_device(layer_size,0);
    thrust::device_vector<float> energy_vector_device(layer_size,0);
    thrust::device_vector<bool> stencil(layer_size);

    /* 4. Storms simulation */
    for(int i=0; i<num_storms; i++) {

        /* 4.1. Add impacts energies to layer cells */
        /* For each particle */
        for(int j=0; j<storms[i].size; j++ ) {
            /* Get impact energy (expressed in thousandths) */
            float energy = (float)storms[i].posval[j*2+1] * 1000;
            /* Get impact position */
            int position = storms[i].posval[j*2]; //TODO check if position is outside of range
            int translated_position = layer_size-position; //relative position to find right part of lookup
            if(translated_position+layer_size > 2*layer_size){
                std::cerr << "position outside of layer" << std::endl;
                exit(EXIT_FAILURE);
            }

            //Update
            // float energy_k = energy / layer_size / atenuacion;
            thrust::transform(thrust::device, //TODO 
                                look_up_device.begin()+translated_position, 
                                look_up_device.begin()+translated_position+layer_size-1,
                                energy_vector_device.begin(),
                                thrust::placeholders::_1*energy
                            );
            // if ( energy_k >= THRESHOLD / layer_size || energy_k <= -THRESHOLD / layer_size )
            thrust::transform(thrust::device, 
                                energy_vector_device.begin(),
                                energy_vector_device.end(),
                                stencil.begin(),
                                above_threshold()
                            );
            // layer[k] = layer[k] + energy_k;
            thrust::transform_if(thrust::device,
                                energy_vector_device.begin(),
                                energy_vector_device.end(),
                                layer_device.begin(),
                                stencil.begin(),
                                layer_device.begin(),
                                thrust::plus<float>(),
                                thrust::identity<bool>()
                            );
        }
        
        /* 4.2. Energy relaxation between storms */
        /* 4.2.1. Copy values to the ancillary array */
        thrust::device_vector<float> layer_copy = layer_device;

        /* 4.2.2. Update layer using the ancillary values.
                  Skip updating the first and last positions */
        // for(int k=1; k<layer_size-1; k++ )
        //     layer[k] = ( layer_copy[k-1] + layer_copy[k] + layer_copy[k+1] ) / 3;
        thrust::transform(thrust::device,
                            layer_copy.begin(), 
                            layer_copy.end()-2, 
                            layer_device.begin()+1, 
                            layer_device.begin()+1,
                            thrust::plus<float>()
                        );
        thrust::transform(thrust::device,
                        layer_copy.begin()+2, 
                        layer_copy.end(), 
                        layer_device.begin()+1, 
                        layer_device.begin()+1,
                        (thrust::placeholders::_1 +
                        thrust::placeholders::_2)/3.0f
                    );
        hipMemcpy(layer, layer_device.data().get(), layer_size*sizeof(float), hipMemcpyDeviceToHost);
        /* 4.3. Locate the maximum value in the layer, and its position */
        thrust::device_vector<float>::iterator result;
        result = thrust::max_element(thrust::device, layer_device.begin()+1, layer_device.end()-1);
        maximum[i] = *result;
        positions[i] = thrust::distance(layer_device.begin(), result);
        // find_local_maximum(layer, layer_size, maximum[i], positions[i]); //TODO delete
    }
    hipMemcpy(layer_device.data().get(), layer, layer_size*sizeof(float), hipMemcpyHostToDevice);
}

/* THIS FUNCTION CAN BE MODIFIED */
/* Function to update a single position of the layer */
void update( float *layer, int layer_size, int k, int pos, float energy ) {
    /* 1. Compute the absolute value of the distance between the
        impact position and the k-th position of the layer */
    int distance = pos - k;
    if ( distance < 0 ) distance = - distance;

    /* 2. Impact cell has a distance value of 1 */
    distance = distance + 1;

    /* 3. Square root of the distance */
    /* NOTE: Real world atenuation typically depends on the square of the distance.
       We use here a tailored equation that affects a much wider range of cells */
    float atenuacion = sqrtf( (float)distance );

    /* 4. Compute attenuated energy */
    float energy_k = energy / layer_size / atenuacion;

    /* 5. Do not add if its absolute value is lower than the threshold */
    if ( energy_k >= THRESHOLD / layer_size || energy_k <= -THRESHOLD / layer_size )
        layer[k] = layer[k] + energy_k;
}


/* ANCILLARY FUNCTIONS: These are not called from the code section which is measured, leave untouched */
/* DEBUG function: Prints the layer status */
void debug_print(int layer_size, float *layer, int *positions, float *maximum, int num_storms ) {
    int i,k;
    /* Only print for array size up to 35 (change it for bigger sizes if needed) */
    if ( layer_size <= 35 ) {
        /* Traverse layer */
        for( k=0; k<layer_size; k++ ) {
            /* Print the energy value of the current cell */
            printf("%10.4f |", layer[k] );

            /* Compute the number of characters. 
               This number is normalized, the maximum level is depicted with 60 characters */
            int ticks = (int)( 60 * layer[k] / maximum[num_storms-1] );

            /* Print all characters except the last one */
            for (i=0; i<ticks-1; i++ ) printf("o");

            /* If the cell is a local maximum print a special trailing character */
            if ( k>0 && k<layer_size-1 && layer[k] > layer[k-1] && layer[k] > layer[k+1] )
                printf("x");
            else
                printf("o");

            /* If the cell is the maximum of any storm, print the storm mark */
            for (i=0; i<num_storms; i++) 
                if ( positions[i] == k ) printf(" M%d", i );

            /* Line feed */
            printf("\n");
        }
    }
}

/*
 * Function: Read data of particle storms from a file
 */
Storm read_storm_file(char *fname ) {
    FILE *fstorm = fopen( fname, "r" );
    if ( fstorm == NULL ) {
        fprintf(stderr,"Error: Opening storm file %s\n", fname );
        exit( EXIT_FAILURE );
    }

    Storm storm;    
    int ok = fscanf(fstorm, "%d", &(storm.size) );
    if ( ok != 1 ) {
        fprintf(stderr,"Error: Reading size of storm file %s\n", fname );
        exit( EXIT_FAILURE );
    }

    storm.posval = (int *)malloc( sizeof(int) * storm.size * 2 );
    if ( storm.posval == NULL ) {
        fprintf(stderr,"Error: Allocating memory for storm file %s, with size %d\n", fname, storm.size );
        exit( EXIT_FAILURE );
    }
    
    int elem;
    for ( elem=0; elem<storm.size; elem++ ) {
        ok = fscanf(fstorm, "%d %d\n", 
                    &(storm.posval[elem*2]),
                    &(storm.posval[elem*2+1]) );
        if ( ok != 2 ) {
            fprintf(stderr,"Error: Reading element %d in storm file %s\n", elem, fname );
            exit( EXIT_FAILURE );
        }
    }
    fclose( fstorm );

    return storm;
}

// // Energy relaxation between storms (moving average filter over windowSize elements)
// void energy_relaxation(thrust::device_vector<float>& layer){
//         /* 4.2. Energy relaxation between storms */
//         /* 4.2.1. Copy values to the ancillary array */
//         thrust::device_vector<float> layer_copy = layer;

//         /* 4.2.2. Update layer using the ancillary values.
//                   Skip updating the first and last positions */
//         // for(int k=1; k<layer_size-1; k++ )
//         //     layer[k] = ( layer_copy[k-1] + layer_copy[k] + layer_copy[k+1] ) / 3;
//         thrust::transform(thrust::device,
//                             layer_copy.begin(), 
//                             layer_copy.end()-2, 
//                             layer.begin()+1, 
//                             layer.begin()+1,
//                             thrust::plus<float>()
//                         );
//         thrust::transform(thrust::device,
//                         layer_copy.begin()+2, 
//                         layer_copy.end(), 
//                         layer.begin()+1, 
//                         layer.begin()+1,
//                         (thrust::placeholders::_1 +
//                         thrust::placeholders::_2)/3.0f
//                     );


// }

void find_local_maximum(float* layer, const int& layer_size, float& maximum, int& position ){
    for(int k=1; k<layer_size-1; k++ ) {
        /* Check it only if it is a local maximum */
        if ( layer[k] > layer[k-1] && layer[k] > layer[k+1] ) {
            if ( layer[k] > maximum ) {
                maximum = layer[k];
                position = k;
            }
        }
    }
}
}; //end of namespace SEQUENTIAL